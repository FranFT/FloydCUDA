#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string.h>
#include <time.h>
#include <algorithm>
#include "Graph.h"

#define blocksize 256

using namespace std;

//**************************************************************************
// Kernel to update the Matrix at k-th iteration
__global__ void floyd_kernel(int * M, const int nverts, const int k)
{

}



int main(int argc, char *argv[])
{
	// Control de errores de la entrada.
	if (argc != 2)
	{
		cerr << "Sintaxis: " << argv[0] << " <archivo de grafo>" << endl;
		return(-1);
	}


	//Get GPU information
	int devID;
	hipDeviceProp_t props;
	hipError_t err;

	// Obtenemos el ID del dispositivo que se va a usar y comprobamos si ha habido
	// alg�n tipo de error.
	err = hipGetDevice(&devID);
	if (err != hipSuccess) { cout << "Error while trying to get device ID." << endl; }

	// Obtenemos las propiedades del dispositivo correspondiente al ID obtenido anteriormente.
	hipGetDeviceProperties(&props, devID);
	printf("Device %d: \"%s\" with Compute %d.%d capability\n",
		devID, props.name, props.major, props.minor);
	cout << "Numero de multiprocesadores: " << props.multiProcessorCount << endl;
	cout << "Numero max. de hebras por multiprocesador: " << props.maxThreadsPerMultiProcessor << endl;
	cout << "Numero hebras por Wrap: " << props.warpSize << endl;
	
	/****
	**
	** Inicializaci�n de los datos del algoritmo
	**
	****/

	// Leemos el fichero de entrada y lo guardamos en G.
	Graph G;
	G.lee(argv[1]);
	//cout << "EL Grafo de entrada es:"<<endl;
	//G.imprime();

	// Obtenemos el n�mero de vertices del problema.
	const int nverts = G.vertices;

	// Fijamos el n�mero de iteraciones del algoritmo.
	const int niters = nverts;

	// Calculamos el n�mero de elementos de la matriz.
	const int nverts2 = nverts*nverts;

	// Reservamos memoria en el HOST para la matriz de salida
	// en base al n�mero de elementos de la matriz.
	int *c_Out_M = new int[nverts2];

	// Calculamos el n�mero de bytes que ocupa la matriz de salida.
	int size = nverts2*sizeof(int);

	// Declaramos el punter� que apuntar� a la zona de memoria en DEVICE
	// que se emplear� para la matriz de entrada.
	int * d_In_M = NULL;



	/****
	**
	** Fase paralela del algoritmo. (Computaci�n en la GPU).
	**
	****/

	// Reservamos un n�mero de bytes de memoria DEVICE para la matriz
	// de entrada igual al n�mero de bytes que ocupa la matriz de salida
	// en memoria HOST.
	err = hipMalloc((void **)&d_In_M, size);
	if (err != hipSuccess) { cout << "ERROR: Bad Allocation in Device Memory" << endl; }

	// Tomamos la primera media del tiempo.
	double  t1 = clock();

	// Copiamos en la matriz alojada en memoria DEVICE la matriz de datos
	// le�da del fichero de entrada en memoria del HOST.
	err = hipMemcpy(d_In_M, G.Get_Matrix(), size, hipMemcpyHostToDevice);
	if (err != hipSuccess) { cout << "ERROR: COPY MATRIX TO DEVICE" << endl; }

	// Bucle principal del algoritmo.
	for (int k = 0; k<niters; k++)
	{

		//*******************************************************************

		//Kernel Launch 



		//*******************************************************************  

		// Comprobamos si se ha producido alg�n error en la iteraci�n k-�sima.
		err = hipGetLastError();

		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to launch kernel!\n");
			exit(EXIT_FAILURE);
		}

	}

	// Cuando finaliza el bucle principal del algoritmo, copiamos en la
	// matriz de salida en memoria HOST, la matriz que se ubica en memoria
	// DEVICE, que contiene la soluci�n.
	hipMemcpy(c_Out_M, d_In_M, size, hipMemcpyDeviceToHost);

	// Tomamos la segunda medida de tiempo.
	double Tgpu = clock();

	// Liberamos la memoria reservada previamente en DEVICE.
	err = hipFree(d_In_M);
	if (err != hipSuccess) { cout << "ERROR: Bad Release in Device Memory" << endl; }

	// Calculamos el tiempo empleado en resolver el problema.
	Tgpu = (Tgpu - t1) / CLOCKS_PER_SEC;
	cout << "Tiempo gastado GPU= " << Tgpu << endl << endl;



	/****
	**
	**	Versi�n secuencial.
	**
	****/

	// Tomamos la primera medida de tiempo.
	t1 = clock();

	// Bucle principal del algoritmo.
	for (int k = 0; k<niters; k++)
		for (int i = 0; i<nverts; i++)
			for (int j = 0; j<nverts; j++)
				if (i != j && i != k && j != k)
				{
					int vikj = min(G.arista(i, k) + G.arista(k, j), G.arista(i, j));
					G.inserta_arista(i, j, vikj);
				}

	// Tomamos la segunda medida de tiempo.
	double t2 = clock();

	// Calculamos el tiempo empleado en resolver el problema
	// secuencial.
	t2 = (t2 - t1) / CLOCKS_PER_SEC;

	//  cout << endl<<"EL Grafo con las distancias de los caminos m�s cortos es:"<<endl<<endl;
	//  G.imprime();
	cout << "Tiempo gastado CPU= " << t2 << endl << endl;
	cout << "Ganancia= " << t2 / Tgpu << endl;

	// Por �ltimo comprobamos que los resultados de la versi�n de CUDA y
	// la versi�n secuencial es la misma.
	for (int i = 0; i<nverts; i++)
		for (int j = 0; j<nverts; j++)
			if (abs(c_Out_M[i*nverts + j] - G.arista(i, j))>0)
				cout << "Error (" << i << "," << j << ")   "
				<< c_Out_M[i*nverts + j] << "..." << G.arista(i, j) << endl;

	// Liberamos la memoria empleada por la matriz de salida.
	delete[] c_Out_M;
}



